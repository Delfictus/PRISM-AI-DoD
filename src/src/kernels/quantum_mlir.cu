#include "hip/hip_runtime.h"
// Quantum MLIR GPU Kernels - Native Complex Number Support
//
// This provides the actual GPU kernels for quantum state evolution
// with first-class complex number support using hipComplex

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <stdio.h>

// Complex number operations using CUDA's native complex support
__device__ __forceinline__ hipDoubleComplex complex_mul(hipDoubleComplex a, hipDoubleComplex b) {
    return hipCmul(a, b);
}

__device__ __forceinline__ hipDoubleComplex complex_add(hipDoubleComplex a, hipDoubleComplex b) {
    return hipCadd(a, b);
}

__device__ __forceinline__ hipDoubleComplex complex_exp(double theta) {
    return make_hipDoubleComplex(cos(theta), sin(theta));
}

// Quantum Hadamard gate kernel
extern "C" __global__ void hadamard_gate_kernel(
    hipDoubleComplex* state,
    int qubit_index,
    int num_qubits
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int state_dim = 1 << num_qubits;

    if (idx >= state_dim / 2) return;

    // Calculate indices for the two states that differ in qubit_index
    int mask = 1 << qubit_index;
    int idx0 = ((idx >> qubit_index) << (qubit_index + 1)) | (idx & ((1 << qubit_index) - 1));
    int idx1 = idx0 | mask;

    // Apply Hadamard transformation
    hipDoubleComplex amp0 = state[idx0];
    hipDoubleComplex amp1 = state[idx1];

    double sqrt2_inv = 0.7071067811865475; // 1/sqrt(2)
    hipDoubleComplex factor = make_hipDoubleComplex(sqrt2_inv, 0.0);

    state[idx0] = hipCmul(factor, hipCadd(amp0, amp1));
    state[idx1] = hipCmul(factor, hipCsub(amp0, amp1));
}

// CNOT gate kernel
extern "C" __global__ void cnot_gate_kernel(
    hipDoubleComplex* state,
    int control_qubit,
    int target_qubit,
    int num_qubits
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int state_dim = 1 << num_qubits;

    if (idx >= state_dim) return;

    int control_mask = 1 << control_qubit;
    int target_mask = 1 << target_qubit;

    // Only flip target if control is |1>
    if ((idx & control_mask) != 0) {
        int flipped_idx = idx ^ target_mask;
        if (idx < flipped_idx) {
            // Swap amplitudes
            hipDoubleComplex temp = state[idx];
            state[idx] = state[flipped_idx];
            state[flipped_idx] = temp;
        }
    }
}

// Time evolution kernel using Trotter-Suzuki decomposition
extern "C" __global__ void time_evolution_kernel(
    hipDoubleComplex* state,
    const hipDoubleComplex* hamiltonian,
    double time_step,
    int dimension,
    int trotter_steps
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dimension) return;

    // Local copy of state amplitude
    hipDoubleComplex local_state = state[idx];

    // Trotter-Suzuki approximation: e^(-iHt) ≈ (e^(-iHt/n))^n
    double dt = time_step / trotter_steps;

    for (int step = 0; step < trotter_steps; step++) {
        hipDoubleComplex new_amplitude = make_hipDoubleComplex(0.0, 0.0);

        // Matrix-vector multiplication: |ψ'> = e^(-iHdt)|ψ>
        for (int j = 0; j < dimension; j++) {
            hipDoubleComplex H_ij = hamiltonian[idx * dimension + j];
            hipDoubleComplex phase = complex_exp(-dt * hipCreal(H_ij));
            hipDoubleComplex contribution = hipCmul(phase, hipCmul(H_ij, state[j]));
            new_amplitude = hipCadd(new_amplitude, contribution);
        }

        local_state = new_amplitude;
    }

    // Write back evolved state
    state[idx] = local_state;
}

// Quantum Fourier Transform kernel
extern "C" __global__ void qft_kernel(
    hipDoubleComplex* state,
    int num_qubits,
    bool inverse
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = 1 << num_qubits;

    if (idx >= N) return;

    extern __shared__ hipDoubleComplex shared_state[];

    // Load state into shared memory
    shared_state[threadIdx.x] = state[idx];
    __syncthreads();

    // QFT is essentially a DFT on quantum amplitudes
    hipDoubleComplex result = make_hipDoubleComplex(0.0, 0.0);
    double sign = inverse ? 1.0 : -1.0;

    for (int k = 0; k < N; k++) {
        double phase = sign * 2.0 * M_PI * idx * k / N;
        hipDoubleComplex twiddle = complex_exp(phase);
        result = hipCadd(result, hipCmul(twiddle, shared_state[k]));
    }

    // Normalize
    double norm = 1.0 / sqrt((double)N);
    state[idx] = hipCmul(make_hipDoubleComplex(norm, 0.0), result);
}

// VQE ansatz kernel for variational quantum algorithms
extern "C" __global__ void vqe_ansatz_kernel(
    hipDoubleComplex* state,
    const double* parameters,
    int num_qubits,
    int num_layers
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int state_dim = 1 << num_qubits;

    if (idx >= state_dim) return;

    // Apply parameterized quantum circuit
    for (int layer = 0; layer < num_layers; layer++) {
        // Rotation layer
        for (int qubit = 0; qubit < num_qubits; qubit++) {
            int param_idx = layer * num_qubits + qubit;
            double theta = parameters[param_idx];

            int mask = 1 << qubit;
            if ((idx & mask) != 0) {
                // Apply Rz rotation
                hipDoubleComplex phase = complex_exp(theta / 2.0);
                state[idx] = hipCmul(state[idx], phase);
            } else {
                // Apply Rz rotation (opposite phase)
                hipDoubleComplex phase = complex_exp(-theta / 2.0);
                state[idx] = hipCmul(state[idx], phase);
            }
        }

        // Entangling layer (linear connectivity)
        __syncthreads();
        for (int qubit = 0; qubit < num_qubits - 1; qubit++) {
            // CNOT between qubit and qubit+1
            int control_mask = 1 << qubit;
            int target_mask = 1 << (qubit + 1);

            if ((idx & control_mask) != 0 && (idx & target_mask) == 0) {
                int flipped_idx = idx | target_mask;
                hipDoubleComplex temp = state[idx];
                state[idx] = state[flipped_idx];
                state[flipped_idx] = temp;
            }
        }
        __syncthreads();
    }
}

// Measurement kernel - calculates probabilities
extern "C" __global__ void measurement_kernel(
    const hipDoubleComplex* state,
    double* probabilities,
    int dimension
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dimension) return;

    hipDoubleComplex amp = state[idx];
    probabilities[idx] = hipCreal(amp) * hipCreal(amp) + hipCimag(amp) * hipCimag(amp);
}

// Helper function to check CUDA errors
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                   hipGetErrorString(error)); \
            return error; \
        } \
    } while(0)

// C interface for calling from Rust
extern "C" {
    // Initialize quantum state to |00...0>
    hipError_t quantum_init_state(hipDoubleComplex* state, int dimension) {
        CHECK_CUDA_ERROR(hipMemset(state, 0, dimension * sizeof(hipDoubleComplex)));
        hipDoubleComplex one = make_hipDoubleComplex(1.0, 0.0);
        CHECK_CUDA_ERROR(hipMemcpy(state, &one, sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        return hipSuccess;
    }

    // Apply Hadamard gate
    hipError_t quantum_hadamard(hipDoubleComplex* state, int qubit, int num_qubits) {
        int state_dim = 1 << num_qubits;
        int num_blocks = (state_dim / 2 + 255) / 256;
        hadamard_gate_kernel<<<num_blocks, 256>>>(state, qubit, num_qubits);
        return hipGetLastError();
    }

    // Apply CNOT gate
    hipError_t quantum_cnot(hipDoubleComplex* state, int control, int target, int num_qubits) {
        int state_dim = 1 << num_qubits;
        int num_blocks = (state_dim + 255) / 256;
        cnot_gate_kernel<<<num_blocks, 256>>>(state, control, target, num_qubits);
        return hipGetLastError();
    }

    // Time evolution
    hipError_t quantum_evolve(
        hipDoubleComplex* state,
        const hipDoubleComplex* hamiltonian,
        double time,
        int dimension,
        int trotter_steps
    ) {
        int num_blocks = (dimension + 255) / 256;
        time_evolution_kernel<<<num_blocks, 256>>>(state, hamiltonian, time, dimension, trotter_steps);
        return hipGetLastError();
    }

    // Quantum Fourier Transform
    hipError_t quantum_qft(hipDoubleComplex* state, int num_qubits, bool inverse) {
        int state_dim = 1 << num_qubits;
        int threads = min(256, state_dim);
        int blocks = (state_dim + threads - 1) / threads;
        size_t shared_mem = threads * sizeof(hipDoubleComplex);
        qft_kernel<<<blocks, threads, shared_mem>>>(state, num_qubits, inverse);
        return hipGetLastError();
    }

    // VQE ansatz
    hipError_t quantum_vqe_ansatz(
        hipDoubleComplex* state,
        const double* parameters,
        int num_qubits,
        int num_layers
    ) {
        int state_dim = 1 << num_qubits;
        int num_blocks = (state_dim + 255) / 256;
        vqe_ansatz_kernel<<<num_blocks, 256>>>(state, parameters, num_qubits, num_layers);
        return hipGetLastError();
    }

    // Measure quantum state
    hipError_t quantum_measure(
        const hipDoubleComplex* state,
        double* probabilities,
        int dimension
    ) {
        int num_blocks = (dimension + 255) / 256;
        measurement_kernel<<<num_blocks, 256>>>(state, probabilities, dimension);
        return hipGetLastError();
    }
}